
#include <hip/hip_runtime.h>
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#include <chrono>
using namespace std;

__global__ void kernel(int dim_m, int dim_n, int dim_k,
		       float *d_a, float *d_b, float *d_c) {
  int offset_a_m = 64 * blockIdx.x;
  int offset_b_n = 64 * blockIdx.y;
  int a_m = threadIdx.x % 8 * 8;
  int a_k = threadIdx.x / 8;
  int b_n = threadIdx.x;

  struct __align__(16) vec_t { float d[8]; };
  vec_t *tile_a;
  vec_t *tile_b;
  vec_t __align__(16) thread_a;
  vec_t __align__(16) thread_b;
  __shared__ float __align__(16) block_a[8][64];
  __shared__ float __align__(16) block_b[8][64];
  float __align__(16) fragment_a[8];
  float __align__(16) fragment_b[8];
  float __align__(16) fragment_c[8][8];

  tile_a = reinterpret_cast<vec_t*>(&d_a[offset_a_m + a_m + a_k * dim_m]);
  tile_b = reinterpret_cast<vec_t*>(&d_b[(offset_b_n + b_n) * dim_k]);
  for (int m = 0; m < 8; ++m)
    for (int n = 0; n < 8; ++n)
      fragment_c[m][n] = 0;

  int warp_id = threadIdx.x / 32;
  int warp_x = 0;
  int warp_y = warp_id;
  int lane_id = threadIdx.x % 32;
  int lane_x = lane_id / 4;
  int lane_y = lane_id % 4;
  int offset_x = warp_x * 64 + lane_x * 8;
  int offset_y = warp_y * 32 + lane_y * 8;
  int offset_a_k = 0;
  int offset_b_k = 0;
  for (int kk = 0; kk < dim_k; kk += 8) {
    thread_a = tile_a[offset_a_k];
    thread_b = tile_b[offset_b_k];
    __syncthreads();
    for (int j = 0; j < 8; ++j) {
      block_a[a_k][a_m + j] = thread_a.d[j];
      block_b[j][b_n] = thread_b.d[j];
    }
    __syncthreads();
    offset_a_k += dim_m;
    offset_b_k ++;
#pragma unroll
    for (int k = 0; k < 8; k++) {
      for (int j = 0; j < 8; ++j) {
	fragment_a[j] = block_a[k][offset_y + j];
	fragment_b[j] = block_b[k][offset_x + j];
      }
      for (int m = 0; m < 8; ++m) {
	for (int n = 0; n < 8; ++n) {
	  fragment_c[m][n] += fragment_a[m] * fragment_b[n];
	}
      }
    }
  }
  for (int j = 0; j < 8; ++j) {
    int tx = offset_x + j;
    int ty = offset_y;
    int bx = 64 * blockIdx.y + tx;
    int by = 64 * blockIdx.x + ty;
    for (int i = 0; i < 8; ++i) {
      if (bx < dim_n && (by + i) < dim_m) {
	d_c[bx * dim_m + by + i] = fragment_c[i][j];
      }
    }
  }
}

int main(int argc, const char **argv) {
  int m = 10240;
  int k = 4096;
  int n = 8192;
  float alpha = 1.0;
  float beta = 0.0;
  int Nt = 10;
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m * k * sizeof(float));
  hipMallocManaged(&B, k * n * sizeof(float));
  hipMallocManaged(&C, m * n * sizeof(float));
  hipMallocManaged(&C2, m * n * sizeof(float));
  for (int i=0; i<m; i++)
    for (int j=0; j<k; j++)
      A[k*i+j] = drand48();
  for (int i=0; i<k; i++)
    for (int j=0; j<n; j++)
      B[n*i+j] = drand48();
  for (int i=0; i<n; i++)
    for (int j=0; j<m; j++)
      C[m*i+j] = C2[m*i+j] = 0;
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSgemm(cublas_handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		m,
		n,
		k,
		&alpha,
		A,
		m,
		B,
		k,
		&beta,
		C,
		m);
    hipDeviceSynchronize();
  }
  auto toc = chrono::steady_clock::now();
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = chrono::duration<double>(toc - tic).count() / Nt;
  double cublas_flops = double(num_flops) / tcublas / 1.0e9;
  int tile = 64;
  dim3 block = dim3(tile);
  dim3 grid = dim3((m+tile-1)/tile, (n+tile-1)/tile);
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    kernel<<< grid, block >>>(m,
			      n,
			      k,
			      A,
			      B,
			      C2);
    hipDeviceSynchronize();
  }
  toc = chrono::steady_clock::now();
  double tcutlass = chrono::duration<double>(toc - tic).count() / Nt;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e9;
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      err += fabs(C[m*i+j] - C2[m*i+j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(C2);
  hipblasDestroy(cublas_handle);
}